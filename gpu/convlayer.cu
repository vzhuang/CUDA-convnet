#include "hip/hip_runtime.h"
#include "layer.h"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <hipblas.h>

#include <iostream>


/**
 * Convolutes images
 */
ConvLayer::ConvLayer(int num_filters_, int filter_size_, int stride_) {
  num_filters = num_filters_;
  filter_size = filter_size_;
  stride = stride_;

  // Create cuBLAS handle for fprop
  hipblasCreate(&handle);
}
ConvLayer::~ConvLayer() {
  hipblasDestroy(handle);
}


// __global__ void ConvLayerStretchWeightsKernel(
//   float * dev_weights_data, 
//   float * dev_stretch_weights_data,
//   int num_images) 
// {
//   int x = threadIdx.x;
//   int y = blockIdx.x;

//   float val = dev_weights_data[y * blockDim.x + x];
//   const int step_size = blockDim.x * gridDim.x;

//   for (int n = 0; n < num_images; n++)
//     dev_stretch_weights_data[n * step_size + x * gridDim.x + y] = val;
// }
__global__ void ConvLayerStretchInputKernel(
  float * dev_input_data, 
  float * dev_stretch_input_data,
  int input_num_images,
  int input_num_channels,
  int input_rows,
  int input_cols,
  int stride, 
  int filter_size) 
{
  int n = blockIdx.z;   // dev_input_->dims.num_images
  int c = blockIdx.y;   // dev_input_->dims.num_channels
  int s = blockIdx.x;   // filter_size * filter_size

  int x = threadIdx.x;  // dev_output->dims.rows
  int y = threadIdx.y;  // dev_output->dims.cols

  int i = x * stride + (s / filter_size);
  int j = y * stride + (s % filter_size);

  // const int stretch_index = n * blockDim.x * blockDim.y * gridDim.x * gridDim.y + x * blockDim.y * gridDim.x + y * gridDim.x + s;
  const int stretch_index = n * blockDim.x * blockDim.y * gridDim.x * gridDim.y + x * blockDim.y * gridDim.x * gridDim.y + y * gridDim.x * gridDim.y + c * gridDim.x + s;
  const int input_index = n * input_num_channels * input_rows * input_cols + c * input_rows * input_cols + i * input_cols + j;

  float val = 0;
  if (i < input_rows && j < input_cols)
    val = dev_input_data[input_index];
  dev_stretch_input_data[stretch_index] = val;
}
__global__ void ConvLayerUnStretchKernel(
  float * dev_stretch_output_data,
  float * dev_output_data, 
  float * dev_biases_data) 
{
  int n = blockIdx.y;
  int x = threadIdx.x;
  int y = threadIdx.y;
  int c = blockIdx.x;

  const int stretch_index = n * blockDim.x * blockDim.y * gridDim.x + x * blockDim.y * gridDim.x + y * gridDim.x + c;
  const int unstretch_index = n * blockDim.x * blockDim.y * gridDim.x + c * blockDim.x * blockDim.y + x * blockDim.y + y;

  dev_output_data[unstretch_index] = dev_stretch_output_data[stretch_index] + dev_biases_data[c];
}
void ConvLayer::fprop(Tensor * dev_input_, Tensor ** dev_output_) {
  // // Stretch weights
  // ConvLayerStretchWeightsKernel<<<num_filters, dev_weights->dims.num_images * filter_size * filter_size>>>(
  //     dev_weights->data, 
  //     dev_stretch_weights->data, 
  //     dev_input_->dims.num_images);
  
  // Stretch input
  dim3 dimGrid(filter_size * filter_size, dev_input_->dims.num_channels, dev_input_->dims.num_images);
  dim3 dimBlock(dev_output->dims.rows, dev_output->dims.cols, 1);
  ConvLayerStretchInputKernel<<<dimGrid, dimBlock>>>(
      dev_input_->data, 
      dev_stretch_input->data, 
      dev_input_->dims.num_images,
      dev_input_->dims.num_channels,
      dev_input_->dims.rows,
      dev_input_->dims.cols,
      stride,
      filter_size);

  // Matrix multiplication
  int m = num_filters;
  int n = dev_output->dims.rows * dev_output->dims.cols;
  int k = dev_input_->dims.num_channels * filter_size * filter_size;
  int lda = m;
  int ldb = k;
  int ldc = m;
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int batchSize = dev_input_->dims.num_images;
  hipblasSgemmBatched(handle, 
                     HIPBLAS_OP_N, HIPBLAS_OP_N, 
                     m, n, k, 
                     &alpha, 
                     (const float **) dev_A, lda, 
                     (const float **) dev_B, ldb, 
                     &beta, 
                     dev_C, ldc,
                     batchSize);

  // Unstretch output + add biases
  dim3 dimGrid2(num_filters, dev_input_->dims.num_images);
  dim3 dimBlock2(dev_output->dims.rows, dev_output->dims.cols);
  ConvLayerUnStretchKernel<<<dimGrid2, dimBlock2>>>(
      dev_stretch_output->data, 
      dev_output->data,
      dev_biases->data);

  // Save input for bprop
  prev_input = dev_input_;

  *dev_output_ = dev_output;
}


/**
 * Propagates errors and does weight updates
 */
__global__ void ConvLayerBpropKernel(
  float * dev_input_grad_data, 
  float * dev_output_grad_data,
  float * dev_weights_grad_data,
  float * dev_biases_grad_data,
  float * dev_input_data,
  float * dev_weights_data,
  int filter_size,
  int stride,
  int input_num_channels,
  int input_rows,
  int input_cols) 
{
  int n = blockIdx.y;
  int f = blockIdx.x;
  int i = threadIdx.x;
  int j = threadIdx.y;

  int output_index = ((n * gridDim.x + f) * blockDim.x + i) * blockDim.y + j;
  float output_grad_value = dev_output_grad_data[output_index];

  int min_i = i * stride;
  int max_i = min_i + filter_size;
  if (input_rows < max_i)
    max_i = input_rows;

  int min_j = j * stride;
  int max_j = min_j + filter_size;
  if (input_cols < max_j)
    max_j = input_cols;

  // Propagate for all values
  for (int i2 = min_i; i2 < max_i; i2++)
    for (int j2 = min_j; j2 < max_j; j2++) 
      for (int channel = 0; channel < input_num_channels; channel++) {
        int input_index = ((n * input_num_channels + channel) * input_rows + i2) * input_cols + j2;
        int weight_index = ((channel * filter_size + i2 - min_i) * filter_size + j2 - min_j) * gridDim.x + f;

        dev_input_grad_data[input_index] += 1; //DEBUG dev_weights_data[weight_index] * output_grad_value;
        // dev_weights_grad_data[weight_index] += dev_input_data[input_index] * output_grad_value;
      }


  // int input_index = ((n * input_num_channels + 0) * input_rows + min_i) * input_cols + min_j;
  // dev_input_grad_data[input_index] = output_grad_value;


  // Normalize weight gradients (divide by n_imgs)
  // biases
}
void ConvLayer::bprop(Tensor ** dev_input_grad_, Tensor * dev_output_grad_, float eta) {
  hipMemset(dev_input_grad->data, 0, dev_input_grad->dims.num_images * dev_input_grad->dims.num_channels * dev_input_grad->dims.rows * dev_input_grad->dims.cols * sizeof(float));

  dim3 dimGrid(num_filters, dev_output_grad_->dims.num_images);
  dim3 dimBlock(dev_output_grad_->dims.rows, dev_output_grad_->dims.cols);
  ConvLayerBpropKernel<<<dimGrid, dimBlock>>>(
      dev_input_grad->data, 
      dev_output_grad_->data, 
      dev_weights_grad->data, 
      dev_biases_grad->data, 
      prev_input->data,
      dev_weights->data,
      filter_size,
      stride,
      dev_input_grad->dims.num_channels, 
      dev_input_grad->dims.rows, 
      dev_input_grad->dims.cols);
  
  // Update weights, biases

  *dev_input_grad_ = dev_input_grad;
}


void ConvLayer::get_output_dims(Dimensions * input_dims, Dimensions * output_dims) {
  output_dims->num_images = input_dims->num_images;
  output_dims->num_channels = num_filters;
  output_dims->rows = (input_dims->rows - filter_size) / stride + 1;
  output_dims->cols = (input_dims->cols - filter_size) / stride + 1;
}


void ConvLayer::init_mem(Dimensions * input_dims) {
  // Parameters
  dev_weights = new Tensor(1, input_dims->num_channels, filter_size * filter_size, num_filters, true);
  dev_biases = new Tensor(1, num_filters, 1, 1, true);

  // Initialization . . .
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, dev_weights->data, input_dims->num_channels * num_filters * filter_size * filter_size);

  // Gradients
  dev_weights_grad = new Tensor(1, input_dims->num_channels, filter_size * filter_size, num_filters, true);
  dev_biases_grad = new Tensor(1, num_filters, 1, 1, true);

  // Output
  Dimensions d;
  get_output_dims(input_dims, &d);
  dev_output = new Tensor(&d, true);
  dev_input_grad = new Tensor(input_dims, true);

  // Input is stretched to (filter_size)^2 x (output rows * output cols)
  // Output is temporarily stored before fixing into row major format
  // CuBLAS is column major rather than row major though, hence the swap
  dev_stretch_input = new Tensor(d.num_images, 1, d.rows * d.cols, input_dims->num_channels * filter_size * filter_size, true);
  dev_stretch_output = new Tensor(&d, true);

  // cuBLAS batch processing
  hipMalloc((void **)&dev_A, sizeof(float *) * input_dims->num_images);
  hipMalloc((void **)&dev_B, sizeof(float *) * input_dims->num_images);
  hipMalloc((void **)&dev_C, sizeof(float *) * input_dims->num_images);
  float **A, **B, **C;
  A = (float **) malloc(sizeof(float *) * input_dims->num_images);
  B = (float **) malloc(sizeof(float *) * input_dims->num_images);
  C = (float **) malloc(sizeof(float *) * input_dims->num_images);
  for (int i = 0; i < input_dims->num_images; i++) {
    A[i] = dev_weights->data;
    B[i] = dev_stretch_input->data + i * d.rows * d.cols * input_dims->num_channels * filter_size * filter_size;
    C[i] = dev_stretch_output->data + i * d.num_channels * d.rows * d.cols;
  }
  hipMemcpy(dev_A, A, sizeof(float *) * input_dims->num_images, hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, sizeof(float *) * input_dims->num_images, hipMemcpyHostToDevice);
  hipMemcpy(dev_C, C, sizeof(float *) * input_dims->num_images, hipMemcpyHostToDevice);
  free(A);
  free(B);
  free(C);
}
void ConvLayer::free_mem() {
  delete dev_weights;
  delete dev_biases;

  delete dev_weights_grad;
  delete dev_biases_grad;

  delete dev_output;
  delete dev_input_grad;

  delete dev_stretch_input;
  delete dev_stretch_output;

  hipFree(dev_A);
  hipFree(dev_B);
  hipFree(dev_C);
}

