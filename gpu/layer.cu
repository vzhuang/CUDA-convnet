#include "hip/hip_runtime.h"
#include "layer.h"

#include <algorithm>
#include <hip/hip_runtime.h>

    #include <stdio.h>

/**
 * Max pooling of size by size region
 */
PoolingLayer::PoolingLayer(int pool_size_, int stride_) {
  pool_size = pool_size_;
  stride = stride_;
}


__global__ void PoolingLayerFpropKernel(
  float * dev_input_data, 
  float * dev_output_data, 
  float * dev_switches_row_data,
  float * dev_switches_col_data,
  int pool_size, 
  int stride,
  int input_num_channels,
  int input_rows,
  int input_cols) 
{
  int n = blockIdx.x;
  int c = blockIdx.y;
  int i = threadIdx.x;
  int j = threadIdx.y;

  int index = ((n * gridDim.y + c) * blockDim.x + i) * blockDim.y + j;

  int min_i = i * stride;
  int max_i = min_i + pool_size;
  if (input_rows < max_i)
    max_i = input_rows;

  int min_j = j * stride;
  int max_j = min_j + pool_size;
  if (input_cols < max_j)
    max_j = input_cols;

  // Find max value over the pooling area
  float max_value = -FLT_MAX;
  int max_row = -1;
  int max_col = -1;
  for (int i2 = min_i; i2 < max_i; i2++)
    for (int j2 = min_j; j2 < max_j; j2++) {
      int index2 = ((n * input_num_channels + c) * input_rows + i2) * input_cols + j2;
      float val = dev_input_data[index2];
      if (val > max_value) {
        max_value = val;
        max_row = i2;
        max_col = j2;
      }
    }
  dev_output_data[index] = max_value;
  dev_switches_row_data[index] = max_row;
  dev_switches_col_data[index] = max_col;
}
void PoolingLayer::fprop(Tensor * dev_input_, Tensor ** dev_output_) {
  dim3 dimGrid(dev_output->dims.num_images, dev_output->dims.num_channels);
  dim3 dimBlock(dev_output->dims.rows, dev_output->dims.cols);

  PoolingLayerFpropKernel<<<dimGrid, dimBlock>>>(
      dev_input_->data, 
      dev_output->data, 
      dev_switches_row->data, 
      dev_switches_col->data, 
      pool_size, 
      stride, 
      dev_input_->dims.num_channels, 
      dev_input_->dims.rows, 
      dev_input_->dims.cols);
  
  *dev_output_ = dev_output;
}

/**
 * Propagates errors through max pooling layer (i.e. to max points in prev layer)
 */
__global__ void PoolingLayerBpropKernel(
  float * dev_input_grad_data, 
  float * dev_output_grad_data,
  float * dev_switches_row_data,
  float * dev_switches_col_data,
  int input_num_channels,
  int input_rows,
  int input_cols) 
{
  int n = blockIdx.x;
  int c = blockIdx.y;
  int i = threadIdx.x;
  int j = threadIdx.y;

  int index = ((n * gridDim.y + c) * blockDim.x + i) * blockDim.y + j;

  int max_row = dev_switches_row_data[index];
  int max_col = dev_switches_col_data[index];
  int index2 = ((n * input_num_channels + c) * input_rows + max_row) * input_cols + max_col;

  dev_input_grad_data[index2] = dev_output_grad_data[index];
}
void PoolingLayer::bprop(Tensor ** dev_input_grad_, Tensor * dev_output_grad_, float eta) {
  hipMemset(dev_input_grad->data, 0, dev_input_grad->dims.num_images * dev_input_grad->dims.num_channels * dev_input_grad->dims.rows * dev_input_grad->dims.cols * sizeof(float));

  dim3 dimGrid(dev_output_grad_->dims.num_images, dev_output_grad_->dims.num_channels);
  dim3 dimBlock(dev_output_grad_->dims.rows, dev_output_grad_->dims.cols);

  PoolingLayerBpropKernel<<<dimGrid, dimBlock>>>(
      dev_input_grad->data, 
      dev_output_grad_->data, 
      dev_switches_row->data, 
      dev_switches_col->data, 
      dev_input_grad->dims.num_channels, 
      dev_input_grad->dims.rows, 
      dev_input_grad->dims.cols);
  
  *dev_input_grad_ = dev_input_grad;
}

void PoolingLayer::get_output_dims(Dimensions * input_dims, Dimensions * output_dims) {
  output_dims->num_images = input_dims->num_images;
  output_dims->num_channels = input_dims->num_channels;
  output_dims->rows = (input_dims->rows - pool_size) / stride + 1;
  output_dims->cols = (input_dims->cols - pool_size) / stride + 1;
}

void PoolingLayer::init_mem(Dimensions * input_dims) {
  Dimensions d;
  get_output_dims(input_dims, &d);
  dev_output = new Tensor(&d, true);
  dev_input_grad = new Tensor(input_dims, true);
  dev_switches_row = new Tensor(&d, true);
  dev_switches_col = new Tensor(&d, true);
}

void PoolingLayer::free_mem() {
  delete dev_output;
  delete dev_input_grad;
  delete dev_switches_row;
  delete dev_switches_col;
}

