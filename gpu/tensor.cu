#include "tensor.h"

#include <cstdlib>
#include <hip/hip_runtime.h>


// Constructors that set dimensions and allocates memory for data array
Tensor::Tensor(int num_images_, int num_channels_, int rows_, int cols_, bool gpu_memory_) {
  dims.num_images = num_images_;
  dims.num_channels = num_channels_;
  dims.rows = rows_;
  dims.cols = cols_;

  int arr_size = dims.num_images * dims.num_channels * dims.rows * dims.cols * sizeof(float);
  if (!gpu_memory_)
    data = (float*) malloc(arr_size);
  else
    hipMalloc((void **)&data, arr_size);

  gpu_memory = gpu_memory_;
}
Tensor::Tensor(Dimensions * dims_, bool gpu_memory_) {
  dims.num_images = dims_->num_images;
  dims.num_channels = dims_->num_channels;
  dims.rows = dims_->rows;
  dims.cols = dims_->cols;

  int arr_size = dims.num_images * dims.num_channels * dims.rows * dims.cols * sizeof(float);
  if (!gpu_memory_)
    data = (float*) malloc(arr_size);
  else
    hipMalloc((void **)&data, arr_size);

  gpu_memory = gpu_memory_;
}

// Destructor frees data array
Tensor::~Tensor() {
  free(data);
}

// Getters + setters for convenience
float Tensor::get(int a, int b, int c, int d) {
  int num_channels = dims.num_channels;
  int rows = dims.rows;
  int cols = dims.cols;

  return data[((a * num_channels + b) * rows + c) * cols + d];
}
void Tensor::set(int a, int b, int c, int d, float val) {
  int num_channels = dims.num_channels;
  int rows = dims.rows;
  int cols = dims.cols;

  data[((a * num_channels + b) * rows + c) * cols + d] = val;
}